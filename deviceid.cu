#include "hip/hip_runtime.h"
#include "deviceid.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N])
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	C[i][j] = A[i][j] + B[i][j];
}

extern "C" void cuda_addm(float A[N][N], float B[N][N], float C[N][N])
{
	// Kernel invocation with one block of N * N * 1 threads
	int numBlocks = 1;
	dim3 threadsPerBlock(N, N);
	MatAdd<<<numBlocks, threadsPerBlock>>>(A, B, C);
}

__global__ void VecAdd(float* A, float* B, float* C)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		C[i] = A[i] + B[i];
}

extern "C" void cuda_add(float* h_A, float* h_B, float* h_C)
{
	size_t size = N * sizeof(float);

	// Allocate vectors in device memory
	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);

	// Copy vectors from host memory to device memory
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// Invoke kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

	// Copy result from device memory to host memory
	// h_C contains the result in host memory
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

__global__ void kernel_multiply(float* matrix, float* vector, float* result, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size)
		result[i] = matrix[i] * vector[i];
}

extern "C" void cuda_multiply(float* d_matrix, float* d_vector, float* d_result, int size)
{
	// Invoke kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	kernel_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_matrix, d_vector, d_result, size);
}

extern "C" void cuda_identify()
{
    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
}
